#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <Windows.h>
#include <inttypes.h>
#include <string.h>

#define BLOCKS				1024
#define THREADS				256  // Cannot be less than 256 because at the beginning of each kernel the first 256 threads in a block copies 256 S-box values from global memory to the shared memory
#define BLOCKSLOG				10
#define THREADSLOG				8
int choice = 0; int trials = 0;
const uint16_t g_awTea3LutA[8] = { 0x92A7, 0xA761, 0x974C, 0x6B8C, 0x29CE, 0x176C, 0x39D4, 0x7463 };
const uint16_t g_awTea3LutB[8] = { 0x9D58, 0xA46D, 0x176C, 0x79C4, 0xC62B, 0xB2C9, 0x4D93, 0x2E93 };
const uint8_t g_abTea3Sbox[256] = {
    0x7D, 0xBF, 0x7B, 0x92, 0xAE, 0x7C, 0xF2, 0x10, 0x5A, 0x0F, 0x61, 0x7A, 0x98, 0x76, 0x07, 0x64,
    0xEE, 0x89, 0xF7, 0xBA, 0xC2, 0x02, 0x0D, 0xE8, 0x56, 0x2E, 0xCA, 0x58, 0xC0, 0xFA, 0x2A, 0x01,
    0x57, 0x6E, 0x3F, 0x4B, 0x9C, 0xDA, 0xA6, 0x5B, 0x41, 0x26, 0x50, 0x24, 0x3E, 0xF8, 0x0A, 0x86,
    0xB6, 0x5C, 0x34, 0xE9, 0x06, 0x88, 0x1F, 0x39, 0x33, 0xDF, 0xD9, 0x78, 0xD8, 0xA8, 0x51, 0xB2,
    0x09, 0xCD, 0xA1, 0xDD, 0x8E, 0x62, 0x69, 0x4D, 0x23, 0x2B, 0xA9, 0xE1, 0x53, 0x94, 0x90, 0x1E,
    0xB4, 0x3B, 0xF9, 0x4E, 0x36, 0xFE, 0xB5, 0xD1, 0xA2, 0x8D, 0x66, 0xCE, 0xB7, 0xC4, 0x60, 0xED,
    0x96, 0x4F, 0x31, 0x79, 0x35, 0xEB, 0x8F, 0xBB, 0x54, 0x14, 0xCB, 0xDE, 0x6B, 0x2D, 0x19, 0x82,
    0x80, 0xAC, 0x17, 0x05, 0xFF, 0xA4, 0xCF, 0xC6, 0x6F, 0x65, 0xE6, 0x74, 0xC8, 0x93, 0xF4, 0x7E,
    0xF3, 0x43, 0x9F, 0x71, 0xAB, 0x9A, 0x0B, 0x87, 0x55, 0x70, 0x0C, 0xAD, 0xCC, 0xA5, 0x44, 0xE7,
    0x46, 0x45, 0x03, 0x30, 0x1A, 0xEA, 0x67, 0x99, 0xDB, 0x4A, 0x42, 0xD7, 0xAA, 0xE4, 0xC2, 0xD5,
    0xF0, 0x77, 0x20, 0xC3, 0x3C, 0x16, 0xB9, 0xE2, 0xEF, 0x6C, 0x3D, 0x1B, 0x22, 0x84, 0x2F, 0x81,
    0x1D, 0xB1, 0x3A, 0xE5, 0x73, 0x40, 0xD0, 0x18, 0xC7, 0x6A, 0x9E, 0x91, 0x48, 0x27, 0x95, 0x72,
    0x68, 0x0E, 0x00, 0xFC, 0xC5, 0x5F, 0xF1, 0xF5, 0x38, 0x11, 0x7F, 0xE3, 0x5E, 0x13, 0xAF, 0x37,
    0xE0, 0x8A, 0x49, 0x1C, 0x21, 0x47, 0xD4, 0xDC, 0xB0, 0xEC, 0x83, 0x28, 0xB8, 0xF6, 0xA7, 0xC9,
    0x63, 0x59, 0xBD, 0x32, 0x85, 0x08, 0xBE, 0xD3, 0xFD, 0x4C, 0x2C, 0xFB, 0xA0, 0xC1, 0x9D, 0xB3,
    0x52, 0x8C, 0x5D, 0x29, 0x6D, 0x04, 0xBC, 0x25, 0x15, 0x8B, 0x12, 0x9B, 0xD6, 0x75, 0xA3, 0x97
};
static uint64_t tea3_compute_iv(uint32_t dwFrameNumbers) {
    uint32_t dwXorred = dwFrameNumbers ^ 0xC43A7D51;
    dwXorred = (dwXorred << 8) | (dwXorred >> 24); // rotate left -> translated to single rol instruction
    uint64_t qwIv = ((uint64_t)dwFrameNumbers << 32) | dwXorred;
    return (qwIv >> 8) | (qwIv << 56); // rotate right
}
__device__ static uint64_t gpu_tea3_compute_iv(uint32_t dwFrameNumbers) {
    uint32_t dwXorred = dwFrameNumbers ^ 0xC43A7D51;
    dwXorred = (dwXorred << 8) | (dwXorred >> 24); // rotate left -> translated to single rol instruction
    uint64_t qwIv = ((uint64_t)dwFrameNumbers << 32) | dwXorred;
    return (qwIv >> 8) | (qwIv << 56); // rotate right
}
static uint8_t tea3_state_word_to_newbyte(uint16_t wSt, const uint16_t* awLut) {
    uint8_t bSt0 = wSt;
    uint8_t bSt1 = wSt >> 8;

    uint8_t bDist;
    uint8_t bOut = 0;

    for (int i = 0; i < 8; i++) {
        // taps on bit 5,6 for bSt0 and bit 5,6 for bSt1
        bDist = ((bSt0 >> 5) & 3) | ((bSt1 >> 3) & 12);
        if (awLut[i] & (1 << bDist)) {
            bOut |= 1 << i;
        }

        // rotate one position
        bSt0 = ((bSt0 >> 1) | (bSt0 << 7));
        bSt1 = ((bSt1 >> 1) | (bSt1 << 7));
    }

    return bOut;
}
__device__ static uint8_t gpu_tea3_state_word_to_newbyte2(uint16_t wSt, const uint32_t* awLut, int warpThreadIndex) {
    uint8_t bSt0 = wSt;
    uint8_t bSt1 = wSt >> 8;
    uint8_t bDist;
    uint8_t bOut = 0;

    for (int i = 0; i < 7; i++) {
        // taps on bit 5,6 for bSt0 and bit 5,6 for bSt1
        bDist = ((bSt0 >> 5) & 3) | ((bSt1 >> 3) & 12);
        if (awLut[32 * i + warpThreadIndex] & (1 << bDist)) {            bOut |= 1 << i;        }
        // rotate one position
        bSt0 = ((bSt0 >> 1) | (bSt0 << 7));
        bSt1 = ((bSt1 >> 1) | (bSt1 << 7));
    }
    // taps on bit 5,6 for bSt0 and bit 5,6 for bSt1
    bDist = ((bSt0 >> 5) & 3) | ((bSt1 >> 3) & 12);
    if (awLut[7 * 32 + warpThreadIndex] & (1 << bDist)) {        bOut |= 1 << 7;    }

    return bOut;
}
__device__ static uint8_t gpu_tea3_state_word_to_newbyte(uint16_t wSt, const uint16_t* awLut) {
    uint8_t bSt0 = wSt;
    uint8_t bSt1 = wSt >> 8;

    uint8_t bDist;
    uint8_t bOut = 0;

    /*   for (int i = 0; i < 8; i++) {
           // taps on bit 5,6 for bSt0 and bit 5,6 for bSt1
           bDist = ((bSt0 >> 5) & 3) | ((bSt1 >> 3) & 12);
           if (awLut[i] & (1 << bDist)) {
               bOut |= 1 << i;
           }
           // rotate one position
           bSt0 = ((bSt0 >> 1) | (bSt0 << 7));
           bSt1 = ((bSt1 >> 1) | (bSt1 << 7));
       }*/
    for (int i = 0; i < 7; i++) {
        // taps on bit 5,6 for bSt0 and bit 5,6 for bSt1
        bDist = ((bSt0 >> 5) & 3) | ((bSt1 >> 3) & 12);
        if (awLut[i] & (1 << bDist)) { bOut |= 1 << i; }
        // rotate one position
        bSt0 = ((bSt0 >> 1) | (bSt0 << 7));
        bSt1 = ((bSt1 >> 1) | (bSt1 << 7));
    }
    // taps on bit 5,6 for bSt0 and bit 5,6 for bSt1
    bDist = ((bSt0 >> 5) & 3) | ((bSt1 >> 3) & 12);
    if (awLut[7] & (1 << bDist)) { bOut |= 1 << 7; }

    return bOut;
}
static uint8_t tea3_reorder_state_byte(uint8_t bStByte) {
    // simple re-ordering of bits
    uint8_t bOut = 0;
    bOut |= ((bStByte << 6) & 0x40);
    bOut |= ((bStByte << 1) & 0x20);
    bOut |= ((bStByte << 2) & 0x98);
    bOut |= ((bStByte >> 4) & 0x04);
    bOut |= ((bStByte >> 3) & 0x01);
    bOut |= ((bStByte >> 6) & 0x02);
    return bOut;
}
__device__ static uint8_t gpu_tea3_reorder_state_byte(uint8_t bStByte) {
    // simple re-ordering of bits
    uint8_t bOut = 0;
    bOut |= ((bStByte << 6) & 0x40);
    bOut |= ((bStByte << 1) & 0x20);
    bOut |= ((bStByte << 2) & 0x98);
    bOut |= ((bStByte >> 4) & 0x04);
    bOut |= ((bStByte >> 3) & 0x01);
    bOut |= ((bStByte >> 6) & 0x02);
    return bOut;
}
void tea3(uint32_t dwFrameNumbers, uint8_t* lpKey, uint32_t dwNumKsBytes, uint8_t* lpKsOut) {
    uint8_t abKeyReg[10];
    uint32_t dwNumSkipRounds = 51;

    // init registers
    uint64_t qwIvReg = tea3_compute_iv(dwFrameNumbers);
    memcpy(abKeyReg, lpKey, 10);

    for (int i = 0; i < dwNumKsBytes; i++) {
        for (int j = 0; j < dwNumSkipRounds; j++) {
            // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
            uint8_t bSboxOut = g_abTea3Sbox[abKeyReg[7] ^ abKeyReg[2]] ^ abKeyReg[0];
            memmove(abKeyReg, abKeyReg + 1, 9);
            abKeyReg[9] = bSboxOut;

            // Step 2: Compute 3 bytes derived from current state
            uint8_t bDerivByte12 = tea3_state_word_to_newbyte((qwIvReg >> 8) & 0xffff, g_awTea3LutA);
            uint8_t bDerivByte56 = tea3_state_word_to_newbyte((qwIvReg >> 40) & 0xffff, g_awTea3LutB);
            uint8_t bReordByte4 = tea3_reorder_state_byte((qwIvReg >> 32) & 0xff);

            // Step 3: Combine current state with state derived values, and xor in key derived sbox output
            uint8_t bNewByte = ((qwIvReg >> 56) ^ bReordByte4 ^ bDerivByte12 ^ bSboxOut) & 0xff;
            uint8_t bMixByte = bDerivByte56;

            // Step 4: Update lfsr: leftshift 8, feed/mix in previously generated bytes
            qwIvReg = ((qwIvReg << 8) ^ ((uint64_t)bMixByte << 40)) | bNewByte;
        }
        lpKsOut[i] = (qwIvReg >> 56);
        dwNumSkipRounds = 19;
    }
}
__global__ void tea3_exhaustive(uint32_t dwFrameNumbers, uint32_t dwNumKsBytes, uint8_t* lpKsOut_d, uint16_t *gpu_awTea3LutA_d, uint16_t* gpu_awTea3LutB_d, uint8_t *gpu_abTea3Sbox_d, int trials, uint64_t* gpu_captured_key) {
    __shared__ uint16_t gpu_awTea3LutA[8];
    __shared__ uint16_t gpu_awTea3LutB[8];
    __shared__ uint8_t gpu_abTea3Sbox[256];
    __shared__ uint8_t lpKsOut[10];
    if (threadIdx.x < 256) {
        gpu_abTea3Sbox[threadIdx.x] = gpu_abTea3Sbox_d[threadIdx.x];
        if (threadIdx.x < 10) lpKsOut[threadIdx.x] = lpKsOut_d[threadIdx.x];
        if (threadIdx.x < 8) {
            gpu_awTea3LutA[threadIdx.x] = gpu_awTea3LutA_d[threadIdx.x];
            gpu_awTea3LutB[threadIdx.x] = gpu_awTea3LutB_d[threadIdx.x];
        }
    }
    __syncthreads();
    uint16_t key_right; // rightmost 16 bits
    uint64_t key_left; // leftmost 64 bits
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
//    threadIndex = threadIndex << (64 - BLOCKSLOG - THREADSLOG);
    uint64_t IV = gpu_tea3_compute_iv(dwFrameNumbers);
    for (uint16_t trial = 0; trial < trials; trial++) {
        uint32_t dwNumSkipRounds = 51;
        uint64_t qwIvReg = IV;
        key_right = trial; // rightmost 16 bits
        key_left = threadIndex; // leftmost 64 bits
        int flag = 1;

        for (int i = 0; (i < dwNumKsBytes) && flag==1; i++) {
            for (int j = 0; j < dwNumSkipRounds; j++) {
                // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
                uint8_t bSboxOut = gpu_abTea3Sbox[(key_left & 0xff) ^ ((key_left >> 40) & 0xff)] ^ (key_left >> 56);
                key_left = (key_left << 8) | (key_right >> 8);
                key_right = (key_right << 8) | bSboxOut;
                // Step 2: Compute 3 bytes derived from current state
                uint8_t bDerivByte12 = gpu_tea3_state_word_to_newbyte((qwIvReg >> 8) & 0xffff, gpu_awTea3LutA);
                uint8_t bDerivByte56 = gpu_tea3_state_word_to_newbyte((qwIvReg >> 40) & 0xffff, gpu_awTea3LutB);
                uint8_t bReordByte4 = gpu_tea3_reorder_state_byte((qwIvReg >> 32) & 0xff);
                // Step 3: Combine current state with state derived values, and xor in key derived sbox output
                uint8_t bNewByte = ((qwIvReg >> 56) ^ bReordByte4 ^ bDerivByte12 ^ bSboxOut) & 0xff;
                uint8_t bMixByte = bDerivByte56;
                // Step 4: Update lfsr: leftshift 8, feed/mix in previously generated bytes
                qwIvReg = ((qwIvReg << 8) ^ ((uint64_t)bMixByte << 40)) | bNewByte;
            }
 //           lpKsOut[i] = (qwIvReg >> 56);
            if ((qwIvReg >> 56) != lpKsOut[i]) flag=0;
//            if (threadIndex == 0 && trial==0) printf("%llx %x\n", qwIvReg >> 56, lpKsOut[i]);
            dwNumSkipRounds = 19;
        }
        if (flag == 1) { gpu_captured_key[0] = threadIndex; gpu_captured_key[1] = trial;    }
//        if (flag == 1) printf("Hello world %llu\n",threadIndex);

    }
}
__global__ void tea3_exhaustive_0conflict(uint32_t dwFrameNumbers, uint32_t dwNumKsBytes, uint8_t* lpKsOut_d, uint16_t* gpu_awTea3LutA_d, uint16_t* gpu_awTea3LutB_d, uint8_t* gpu_abTea3Sbox_d, int trials, uint64_t* gpu_captured_key) {
    __shared__ uint32_t gpu_awTea3LutA[256];
    __shared__ uint32_t gpu_awTea3LutB[256];
    __shared__ uint32_t gpu_abTea3Sbox[256][32];
    __shared__ uint8_t lpKsOut[10];
    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 256) {
        for (int i = 0; i < 32; i++) gpu_abTea3Sbox[threadIdx.x][i] = gpu_abTea3Sbox_d[threadIdx.x];
        if (threadIdx.x < 10) lpKsOut[threadIdx.x] = lpKsOut_d[threadIdx.x];
        if (threadIdx.x < 8) {
            for (int i = 0; i < 32; i++) {
                gpu_awTea3LutA[32 * threadIdx.x + i] = gpu_awTea3LutA_d[threadIdx.x];
                gpu_awTea3LutB[32 * threadIdx.x + i] = gpu_awTea3LutB_d[threadIdx.x];
            }
        }
    }
    __syncthreads();  
    uint16_t key_right; // rightmost 16 bits
    uint64_t key_left; // leftmost 64 bits
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint64_t IV = gpu_tea3_compute_iv(dwFrameNumbers);

    for (int trial = 0; trial < trials; trial++) {
        uint32_t dwNumSkipRounds = 51;
        uint64_t qwIvReg = IV;
        key_right = trial; // rightmost 16 bits
        key_left = threadIndex; // leftmost 64 bits
        int flag = 1;
        for (int i = 0; (i < dwNumKsBytes) && flag == 1; i++) {
            for (int j = 0; j < dwNumSkipRounds; j++) {
                // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
                uint8_t bSboxOut = gpu_abTea3Sbox[(key_left & 0xff) ^ ((key_left >> 40) & 0xff)][warpThreadIndex] ^ (key_left >> 56);
                key_left = (key_left << 8) ^ (key_right >> 8);
                key_right = (key_right << 8) ^ bSboxOut;
                // Step 2: Compute 3 bytes derived from current state
                uint8_t bDerivByte12 = gpu_tea3_state_word_to_newbyte2((qwIvReg >> 8) & 0xffff, gpu_awTea3LutA,  warpThreadIndex);
                uint8_t bDerivByte56 = gpu_tea3_state_word_to_newbyte2((qwIvReg >> 40) & 0xffff, gpu_awTea3LutB, warpThreadIndex);
                uint8_t bReordByte4 = gpu_tea3_reorder_state_byte((qwIvReg >> 32) & 0xff);
                // Step 3: Combine current state with state derived values, and xor in key derived sbox output
                uint8_t bNewByte = ((qwIvReg >> 56) ^ bReordByte4 ^ bDerivByte12 ^ bSboxOut) & 0xff;
                uint8_t bMixByte = bDerivByte56;
                // Step 4: Update lfsr: leftshift 8, feed/mix in previously generated bytes
                qwIvReg = ((qwIvReg << 8) ^ ((uint64_t)bMixByte << 40)) | bNewByte;
            }
            if ((qwIvReg >> 56) != lpKsOut[i]) flag = 0;
            dwNumSkipRounds = 19;
        }
        if (flag == 1) { gpu_captured_key[0] = threadIndex; gpu_captured_key[1] = trial; }
    }
}
__global__ void tea3_exhaustive_1conflict(uint32_t dwFrameNumbers, uint32_t dwNumKsBytes, uint8_t* lpKsOut_d, uint16_t* gpu_awTea3LutA_d, uint16_t* gpu_awTea3LutB_d, uint8_t* gpu_abTea3Sbox_d, int trials, uint64_t* gpu_captured_key) {
    __shared__ uint32_t gpu_awTea3LutA[256];
    __shared__ uint32_t gpu_awTea3LutB[256];
    __shared__ uint8_t gpu_abTea3Sbox[256];
    __shared__ uint8_t lpKsOut[10];
    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 256) {
        gpu_abTea3Sbox[threadIdx.x] = gpu_abTea3Sbox_d[threadIdx.x];
        if (threadIdx.x < 10) lpKsOut[threadIdx.x] = lpKsOut_d[threadIdx.x];
        if (threadIdx.x < 8) {
            for (int i = 0; i < 32; i++) {
                gpu_awTea3LutA[32 * threadIdx.x + i] = gpu_awTea3LutA_d[threadIdx.x];
                gpu_awTea3LutB[32 * threadIdx.x + i] = gpu_awTea3LutB_d[threadIdx.x];
            }
        }
    }
    __syncthreads();
    uint16_t key_right; // rightmost 16 bits
    uint64_t key_left; // leftmost 64 bits
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint64_t IV = gpu_tea3_compute_iv(dwFrameNumbers);
    for (int trial = 0; trial < trials; trial++) {
        uint64_t qwIvReg = IV;
        uint32_t dwNumSkipRounds = 51;
        key_right = trial; // rightmost 16 bits
        key_left = threadIndex; // leftmost 64 bits
        int flag = 1;
        for (int i = 0; (i < dwNumKsBytes) && flag == 1; i++) {
            for (int j = 0; j < dwNumSkipRounds; j++) {
                // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
                uint8_t bSboxOut = gpu_abTea3Sbox[(key_left & 0xff) ^ ((key_left >> 40) & 0xff)] ^ (key_left >> 56);
                key_left = (key_left << 8) ^ (key_right >> 8);
                key_right = (key_right << 8) ^ bSboxOut;
                // Step 2: Compute 3 bytes derived from current state
                uint8_t bDerivByte12 = gpu_tea3_state_word_to_newbyte2((qwIvReg >> 8) & 0xffff, gpu_awTea3LutA, warpThreadIndex);
                uint8_t bDerivByte56 = gpu_tea3_state_word_to_newbyte2((qwIvReg >> 40) & 0xffff, gpu_awTea3LutB, warpThreadIndex);
                uint8_t bReordByte4 = gpu_tea3_reorder_state_byte((qwIvReg >> 32) & 0xff);
                // Step 3: Combine current state with state derived values, and xor in key derived sbox output
                uint8_t bNewByte = ((qwIvReg >> 56) ^ bReordByte4 ^ bDerivByte12 ^ bSboxOut) & 0xff;
                uint8_t bMixByte = bDerivByte56;
                // Step 4: Update lfsr: leftshift 8, feed/mix in previously generated bytes
                qwIvReg = ((qwIvReg << 8) ^ ((uint64_t)bMixByte << 40)) | bNewByte;
            }
            if ((qwIvReg >> 56) != lpKsOut[i]) flag = 0;
            dwNumSkipRounds = 19;
        }
        if (flag == 1) { gpu_captured_key[0] = threadIndex; gpu_captured_key[1] = trial; }
    }
}
// tea3_exhaustive_bitsliced kernel keeps each bit of the state in a byte (8-bit) variable
// These 8-bit variables keep 8 different states so one encryption performed by a thread actually means 8 encryptions in parallel
__global__ void tea3_exhaustive_bitsliced(uint8_t *reg, uint32_t dwNumKsBytes, uint8_t* lpKsOut_d, uint8_t* gpu_abTea3Sbox_d, int trials, uint64_t* gpu_captured_key) {
    __shared__ uint8_t gpu_abTea3Sbox[256];
    __shared__ uint8_t lpKsOut[80];
    if (threadIdx.x < 256) {
        gpu_abTea3Sbox[threadIdx.x] = gpu_abTea3Sbox_d[threadIdx.x];
        if (threadIdx.x < 80) lpKsOut[threadIdx.x] = lpKsOut_d[threadIdx.x];
    }
    __syncthreads();
    uint16_t key_right[8]; // rightmost 16 bits
    uint64_t key_left[8]; // leftmost 64 bits
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint8_t r[64];
    uint8_t bDerivByte12[8], bDerivByte56[8], bReordByte4[8], bNewByte[8];
    int flag[8];
//    if (threadIndex == 0) for (int i = 0; i < 3; i++) printf("Reg %d: %x\n", i, reg[i]);
    //   threadIndex = threadIndex << (64 - BLOCKSLOG - THREADSLOG);    
    for (int trial = 0; trial < trials; trial++) {
        uint32_t dwNumSkipRounds = 51;
        for (int i = 0; i < 64; i++) r[i] = reg[i];
        for (uint64_t i = 0; i < 8; i++) {
            key_right[i] = trial; // rightmost 16 bits
            key_left[i] = (threadIndex << (64 - BLOCKSLOG - THREADSLOG)) ^ (i<<16); // leftmost 64 bits
            flag[i] = 1;
        }
        int flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7];
        for (int w = 0; (w < dwNumKsBytes) && flag_overall == 1; w++) {
            for (int j = 0; j < dwNumSkipRounds; j++) {
                // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
                uint8_t bSboxOut[8];
                for (int i = 0; i < 8; i++) {
                    bSboxOut[i] = gpu_abTea3Sbox[(key_left[i] & 0xff) ^ ((key_left[i] >> 40) & 0xff)] ^ (key_left[i] >> 56);
                    key_left[i] = (key_left[i] << 8) ^ (key_right[i] >> 8);
                    key_right[i] = (key_right[i] << 8) ^ bSboxOut[i];
                }
 //               if (threadIndex == 0) for (int i = 0; i < 8; i++) printf("%llx %x\n",key_left[i], key_right[i]); // keys are generated correctly 
 //               if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 64; i++) printf("%02x\n", reg[i]); printf("\n"); }
                // Step 2: Compute 3 bytes derived from current state
/*                bDerivByte12[7] = r[45] & r[46] & r[53] ^ r[45] & r[46] ^ r[45] & r[53] & r[54] ^ r[45] & r[53] ^ r[45] & r[54] ^ r[46] & r[53] & r[54] ^ r[53] ^ r[54] ^ 0xff;
                bDerivByte12[6] = r[46] & r[47] & r[54] ^ r[46] & r[47] ^ r[46] & r[54] & r[55] ^ r[46] & r[55] ^ r[46] ^ r[47] & r[55] ^ r[47] ^ r[54] ^ 0xff;
                bDerivByte12[5] = r[40] & r[47] & r[48] ^ r[40] & r[47] & r[55] ^ r[40] & r[48] & r[55] ^ r[40] & r[48] ^ r[40] ^ r[47] & r[48] & r[55] ^ r[48];
                bDerivByte12[4] = r[40] & r[41] & r[48] ^ r[40] & r[41] & r[49] ^ r[40] & r[48] & r[49] ^ r[41] & r[48] & r[49] ^ r[41] & r[48] ^ r[41] ^ r[48] & r[49] ^ r[49];
                bDerivByte12[3] = r[41] & r[42] & r[49] ^ r[41] & r[42] & r[50] ^ r[41] & r[42] ^ r[41] & r[49] & r[50] ^ r[41] & r[49] ^ r[41] ^ r[42] & r[49] & r[50] ^ r[42] ^ r[49] & r[50] ^ r[50];
                bDerivByte12[2] = r[42] & r[43] & r[51] ^ r[42] & r[50] ^ r[43] & r[50] & r[51] ^ r[43] & r[51] ^ r[43] ^ r[51];
                bDerivByte12[1] = r[43] & r[44] & r[52] ^ r[43] & r[44] ^ r[43] & r[51] ^ r[43] & r[52] ^ r[44] & r[51] & r[52] ^ r[44] & r[51] ^ r[44] ^ r[51] & r[52] ^ r[51] ^ r[52];
                bDerivByte12[0] = r[44] & r[45] & r[53] ^ r[44] & r[52] & r[53] ^ r[44] & r[52] ^ r[45] & r[52] & r[53] ^ r[45] ^ r[52] ^ r[53] ^ 0xff;*/ //old

                bDerivByte12[7] = r[50] & r[49] & r[42] ^ r[50] & r[49] ^ r[50] & r[42] & r[41] ^ r[50] & r[42] ^ r[50] & r[41] ^ r[49] & r[42] & r[41] ^ r[42] ^ r[41] ^ 0xff;
                bDerivByte12[6] = r[49] & r[48] & r[41] ^ r[49] & r[48] ^ r[49] & r[41] & r[40] ^ r[49] & r[40] ^ r[49] ^ r[48] & r[40] ^ r[48] ^ r[41] ^ 0xff;
                bDerivByte12[5] = r[55] & r[48] & r[47] ^ r[55] & r[48] & r[40] ^ r[55] & r[47] & r[40] ^ r[55] & r[47] ^ r[55] ^ r[48] & r[47] & r[40] ^ r[47];
                bDerivByte12[4] = r[55] & r[54] & r[47] ^ r[55] & r[54] & r[46] ^ r[55] & r[47] & r[46] ^ r[54] & r[47] & r[46] ^ r[54] & r[47] ^ r[54] ^ r[47] & r[46] ^ r[46];
                bDerivByte12[3] = r[54] & r[53] & r[46] ^ r[54] & r[53] & r[45] ^ r[54] & r[53] ^ r[54] & r[46] & r[45] ^ r[54] & r[46] ^ r[54] ^ r[53] & r[46] & r[45] ^ r[53] ^ r[46] & r[45] ^ r[45];
                bDerivByte12[2] = r[53] & r[52] & r[44] ^ r[53] & r[45] ^ r[52] & r[45] & r[44] ^ r[52] & r[44] ^ r[52] ^ r[44];
                bDerivByte12[1] = r[52] & r[51] & r[43] ^ r[52] & r[51] ^ r[52] & r[44] ^ r[52] & r[43] ^ r[51] & r[44] & r[43] ^ r[51] & r[44] ^ r[51] ^ r[44] & r[43] ^ r[44] ^ r[43];
                bDerivByte12[0] = r[51] & r[50] & r[42] ^ r[51] & r[43] & r[42] ^ r[51] & r[43] ^ r[50] & r[43] & r[42] ^ r[50] ^ r[43] ^ r[42] ^ 0xff;

//                uint8_t bDerivByte12 = gpu_tea3_state_word_to_newbyte((qwIvReg >> 8) & 0xffff, gpu_awTea3LutA);  // F32
//                uint8_t bDerivByte56 = gpu_tea3_state_word_to_newbyte((qwIvReg >> 40) & 0xffff, gpu_awTea3LutB); // F31
/*                bDerivByte56[7] = r[13] & r[14] & r[21] ^ r[13] & r[14] ^ r[13] & r[21] & r[22] ^ r[13] & r[21] ^ r[13] & r[22] ^ r[14] & r[21] & r[22] ^ r[21] & r[22] ^ r[21] ^ r[22];
                bDerivByte56[6] = r[14] & r[15] & r[22] ^ r[14] & r[15] ^ r[14] & r[22] & r[23] ^ r[14] & r[23] ^ r[14] ^ r[15] & r[22] ^ r[15] & r[23] ^ r[22] & r[23] ^ r[22] ^ r[23] ^ 0xff;
                bDerivByte56[5] = r[8] & r[15] & r[16] ^ r[8] & r[16] & r[23] ^ r[8] & r[16] ^ r[8] ^ r[15] & r[23] ^ r[16];
                bDerivByte56[4] = r[8] & r[9] & r[16] ^ r[8] & r[9] & r[17] ^ r[8] & r[9] ^ r[8] & r[16] & r[17] ^ r[8] & r[17] ^ r[9] & r[16] & r[17] ^ r[9] ^ r[17];
                bDerivByte56[3] = r[9] & r[10] & r[18] ^ r[9] & r[10] ^ r[9] & r[17] ^ r[9] & r[18] ^ r[10] & r[17] & r[18] ^ r[10] & r[17] ^ r[10] ^ r[17] & r[18] ^ r[17] ^ r[18] ^ 0xff;
                bDerivByte56[2] = r[10] & r[11] & r[19] ^ r[10] & r[18] ^ r[10] ^ r[11] & r[18] & r[19] ^ r[11] & r[19] ^ r[11] ^ r[18] ^ r[19] ^ 0xff;
                bDerivByte56[1] = r[11] & r[12] & r[20] ^ r[11] & r[19] ^ r[11] & r[20] ^ r[12] & r[19] & r[20] ^ r[12] & r[20] ^ r[12] ^ r[19] & r[20] ^ 0xff;
                bDerivByte56[0] = r[12] & r[13] & r[21] ^ r[12] & r[20] & r[21] ^ r[12] & r[20] ^ r[12] & r[21] ^ r[13] & r[20] & r[21] ^ r[13] ^ r[21] ^ 0xff;*/ //old


                bDerivByte56[7] = r[18] & r[17] & r[10] ^ r[18] & r[17] ^ r[18] & r[10] & r[9] ^ r[18] & r[10] ^ r[18] & r[9] ^ r[17] & r[10] & r[9] ^ r[10] & r[9] ^ r[10] ^ r[9];
                bDerivByte56[6] = r[17] & r[16] & r[9] ^ r[17] & r[16] ^ r[17] & r[9] & r[8] ^ r[17] & r[8] ^ r[17] ^ r[16] & r[9] ^ r[16] & r[8] ^ r[9] & r[8] ^ r[9] ^ r[8] ^ 0xff;
                bDerivByte56[5] = r[23] & r[16] & r[15] ^ r[23] & r[15] & r[8] ^ r[23] & r[15] ^ r[23] ^ r[16] & r[8] ^ r[15];
                bDerivByte56[4] = r[23] & r[22] & r[15] ^ r[23] & r[22] & r[14] ^ r[23] & r[22] ^ r[23] & r[15] & r[14] ^ r[23] & r[14] ^ r[22] & r[15] & r[14] ^ r[22] ^ r[14];
                bDerivByte56[3] = r[22] & r[21] & r[13] ^ r[22] & r[21] ^ r[22] & r[14] ^ r[22] & r[13] ^ r[21] & r[14] & r[13] ^ r[21] & r[14] ^ r[21] ^ r[14] & r[13] ^ r[14] ^ r[13] ^ 0xff;
                bDerivByte56[2] = r[21] & r[20] & r[12] ^ r[21] & r[13] ^ r[21] ^ r[20] & r[13] & r[12] ^ r[20] & r[12] ^ r[20] ^ r[13] ^ r[12] ^ 0xff;
                bDerivByte56[1] = r[20] & r[19] & r[11] ^ r[20] & r[12] ^ r[20] & r[11] ^ r[19] & r[12] & r[11] ^ r[19] & r[11] ^ r[19] ^ r[12] & r[11] ^ 0xff;
                bDerivByte56[0] = r[19] & r[18] & r[10] ^ r[19] & r[11] & r[10] ^ r[19] & r[11] ^ r[19] & r[10] ^ r[18] & r[11] & r[10] ^ r[18] ^ r[10] ^ 0xff;


//                uint8_t bReordByte4 = gpu_tea3_reorder_state_byte((qwIvReg >> 32) & 0xff);
                bReordByte4[0] = r[26];
                bReordByte4[1] = r[31];
                bReordByte4[2] = r[27];
                bReordByte4[3] = r[29];
                bReordByte4[4] = r[30];
                bReordByte4[5] = r[25];
                bReordByte4[6] = r[24];
                bReordByte4[7] = r[28];
 //               if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bDerivByte12[i]); printf("\n");}
  //              if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bDerivByte56[i]); printf("\n"); }
  //              if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bReordByte4[i]); printf("\n"); }
                // Transpose the S-box output
                int k;
                unsigned m, t;
                m = 0x0000000F;

                for (int l = 4; l != 0; l = l >> 1, m = m ^ (m << l)) {
                    for (k = 0; k < 8; k = (k + l + 1) & ~l) {
                        t = (bSboxOut[k] ^ (bSboxOut[k + l] >> l)) & m;
                        bSboxOut[k] = bSboxOut[k] ^ t;
                        bSboxOut[k + l] = bSboxOut[k + l] ^ (t << l);
                    }
                }

                // Transpose the S-box (my implementation)
 /*               uint8_t bSboxOut2[8] = { 0 };
                for (int i = 0; i < 8; i++) {
                    for (int j = 7; j > 0; j--) {
                        bSboxOut2[i] |= bSboxOut[j] & (0x80 >> i);
                        bSboxOut2[i] >>= 1;
                    }
                    bSboxOut2[i] |= bSboxOut[0] & (0x80 >> i);
                }*/

 //               if (threadIndex == 0) for (int i = 0; i < 8; i++) printf("%02x\n", bSboxOut[i]); // Sboxes are transposed correctly
                // Step 3: Combine current state with state derived values, and xor in key derived sbox output
//                uint8_t bNewByte = ((qwIvReg >> 56) ^ bReordByte4 ^ bDerivByte12 ^ bSboxOut) & 0xff;
/*                bNewByte[0] = r[0] ^ bReordByte4[0] ^ bDerivByte12[0] ^ bSboxOut[0]; // bSboxOut must be turned into bSboxOut[8]
                bNewByte[1] = r[1] ^ bReordByte4[1] ^ bDerivByte12[1] ^ bSboxOut[1]; // bSboxOut must be turned into bSboxOut[8]
                bNewByte[2] = r[2] ^ bReordByte4[2] ^ bDerivByte12[2] ^ bSboxOut[2]; // bSboxOut must be turned into bSboxOut[8]
                bNewByte[3] = r[3] ^ bReordByte4[3] ^ bDerivByte12[3] ^ bSboxOut[3]; // bSboxOut must be turned into bSboxOut[8]
                bNewByte[4] = r[4] ^ bReordByte4[4] ^ bDerivByte12[4] ^ bSboxOut[4]; // bSboxOut must be turned into bSboxOut[8]
                bNewByte[5] = r[5] ^ bReordByte4[5] ^ bDerivByte12[5] ^ bSboxOut[5]; // bSboxOut must be turned into bSboxOut[8]
                bNewByte[6] = r[6] ^ bReordByte4[6] ^ bDerivByte12[6] ^ bSboxOut[6]; // bSboxOut must be turned into bSboxOut[8]
                bNewByte[7] = r[7] ^ bReordByte4[7] ^ bDerivByte12[7] ^ bSboxOut[7]; // bSboxOut must be turned into bSboxOut[8]
*/
                for (int i = 0; i < 8; i++) bNewByte[i] = r[i] ^ bReordByte4[i] ^ bDerivByte12[i] ^ bSboxOut[i];



 //               uint8_t bMixByte = bDerivByte56;
                // Step 4: Update lfsr: leftshift 8, feed/mix in previously generated bytes
 //               qwIvReg = ((qwIvReg << 8) ^ ((uint64_t)bMixByte << 40)) | bNewByte;

                r[0] = r[8];
                r[1] = r[9];
                r[2] = r[10];
                r[3] = r[11];
                r[4] = r[12];
                r[5] = r[13];
                r[6] = r[14];
                r[7] = r[15];
                r[8] = r[16];
                r[9] = r[17];
                r[10] = r[18];
                r[11] = r[19];
                r[12] = r[20];
                r[13] = r[21];
                r[14] = r[22];
                r[15] = r[23];
                r[16] = r[24] ^ bDerivByte56[0];
                r[17] = r[25] ^ bDerivByte56[1];
                r[18] = r[26] ^ bDerivByte56[2];
                r[19] = r[27] ^ bDerivByte56[3];
                r[20] = r[28] ^ bDerivByte56[4];
                r[21] = r[29] ^ bDerivByte56[5];
                r[22] = r[30] ^ bDerivByte56[6];
                r[23] = r[31] ^ bDerivByte56[7];
                r[24] = r[32];
                r[25] = r[33];
                r[26] = r[34];
                r[27] = r[35];
                r[28] = r[36];
                r[29] = r[37];
                r[30] = r[38];
                r[31] = r[39];
                r[32] = r[40];
                r[33] = r[41];
                r[34] = r[42];
                r[35] = r[43];
                r[36] = r[44];
                r[37] = r[45];
                r[38] = r[46];
                r[39] = r[47];
                r[40] = r[48];
                r[41] = r[49];
                r[42] = r[50];
                r[43] = r[51];
                r[44] = r[52];
                r[45] = r[53];
                r[46] = r[54];
                r[47] = r[55];
                r[48] = r[56];
                r[49] = r[57];
                r[50] = r[58];
                r[51] = r[59];
                r[52] = r[60];
                r[53] = r[61];
                r[54] = r[62];
                r[55] = r[63];

                r[56] = bNewByte[0];
                r[57] = bNewByte[1];
                r[58] = bNewByte[2];
                r[59] = bNewByte[3];
                r[60] = bNewByte[4];
                r[61] = bNewByte[5];
                r[62] = bNewByte[6];
                r[63] = bNewByte[7];

            }
            //           lpKsOut[i] = (qwIvReg >> 56);
 //           if ((qwIvReg >> 56) != lpKsOut[i]) flag = 0;
            for (int i = 0; i < 8; i++)
                for (int c = 0; c < 8; c++)
                    if ((r[c] & (0x80>>i)) != (lpKsOut[w*8+c] & (0x80 >> i))) flag[i] = 0;
            //            if (threadIndex == 0 && trial==0) printf("%llx %x\n", qwIvReg >> 56, lpKsOut[i]);
            dwNumSkipRounds = 19;
            flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7];
        }
//        if (threadIndex == 0) for (int i = 0; i < 8;i++) printf("%x\n", r[i]);
        if (flag_overall == 1) { gpu_captured_key[0] = threadIndex; gpu_captured_key[1] = trial; }
        //        if (flag == 1) printf("Hello world %llu\n",threadIndex);

    }
}
__global__ void tea3_exhaustive_bitsliced_shared(uint8_t* reg, uint32_t dwNumKsBytes, uint8_t* lpKsOut_d, uint8_t* gpu_abTea3Sbox_d, int trials, uint64_t* gpu_captured_key) {
    __shared__ uint32_t gpu_abTea3Sbox[256][32];
    __shared__ uint8_t lpKsOut[80];
    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 256) {
        for (int i = 0; i < 32; i++) gpu_abTea3Sbox[threadIdx.x][i] = gpu_abTea3Sbox_d[threadIdx.x];
        if (threadIdx.x < 80) lpKsOut[threadIdx.x] = lpKsOut_d[threadIdx.x];
    }
    __syncthreads();
    uint16_t key_right[8]; // rightmost 16 bits
    uint64_t key_left[8]; // leftmost 64 bits
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint8_t r[64];
    uint8_t bDerivByte12[8], bDerivByte56[8], bReordByte4[8], bNewByte[8];
    int flag[8];
    //    if (threadIndex == 0) for (int i = 0; i < 3; i++) printf("Reg %d: %x\n", i, reg[i]);
        //   threadIndex = threadIndex << (64 - BLOCKSLOG - THREADSLOG);    
    for (int trial = 0; trial < trials; trial++) {
        uint32_t dwNumSkipRounds = 51;
        for (int i = 0; i < 64; i++) r[i] = reg[i];
        for (uint64_t i = 0; i < 8; i++) {
            key_right[i] = trial; // rightmost 16 bits
            key_left[i] = (threadIndex << (64 - BLOCKSLOG - THREADSLOG)) ^ (i << 36); // leftmost 64 bits
            flag[i] = 1;
        }
        int flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7];
        for (int w = 0; (w < dwNumKsBytes) && flag_overall == 1; w++) {
            for (int j = 0; j < dwNumSkipRounds; j++) {
                // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
                uint8_t bSboxOut[8];
                for (int i = 0; i < 8; i++) {
                    bSboxOut[i] = gpu_abTea3Sbox[(key_left[i] & 0xff) ^ ((key_left[i] >> 40) & 0xff)][warpThreadIndex] ^ (key_left[i] >> 56);
                    key_left[i] = (key_left[i] << 8) ^ (key_right[i] >> 8);
                    key_right[i] = (key_right[i] << 8) ^ bSboxOut[i];
                }
                //               if (threadIndex == 0) for (int i = 0; i < 8; i++) printf("%llx %x\n",key_left[i], key_right[i]); // keys are generated correctly 
                //               if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 64; i++) printf("%02x\n", reg[i]); printf("\n"); }
                               // Step 2: Compute 3 bytes derived from current state
               /*                bDerivByte12[7] = r[45] & r[46] & r[53] ^ r[45] & r[46] ^ r[45] & r[53] & r[54] ^ r[45] & r[53] ^ r[45] & r[54] ^ r[46] & r[53] & r[54] ^ r[53] ^ r[54] ^ 0xff;
                               bDerivByte12[6] = r[46] & r[47] & r[54] ^ r[46] & r[47] ^ r[46] & r[54] & r[55] ^ r[46] & r[55] ^ r[46] ^ r[47] & r[55] ^ r[47] ^ r[54] ^ 0xff;
                               bDerivByte12[5] = r[40] & r[47] & r[48] ^ r[40] & r[47] & r[55] ^ r[40] & r[48] & r[55] ^ r[40] & r[48] ^ r[40] ^ r[47] & r[48] & r[55] ^ r[48];
                               bDerivByte12[4] = r[40] & r[41] & r[48] ^ r[40] & r[41] & r[49] ^ r[40] & r[48] & r[49] ^ r[41] & r[48] & r[49] ^ r[41] & r[48] ^ r[41] ^ r[48] & r[49] ^ r[49];
                               bDerivByte12[3] = r[41] & r[42] & r[49] ^ r[41] & r[42] & r[50] ^ r[41] & r[42] ^ r[41] & r[49] & r[50] ^ r[41] & r[49] ^ r[41] ^ r[42] & r[49] & r[50] ^ r[42] ^ r[49] & r[50] ^ r[50];
                               bDerivByte12[2] = r[42] & r[43] & r[51] ^ r[42] & r[50] ^ r[43] & r[50] & r[51] ^ r[43] & r[51] ^ r[43] ^ r[51];
                               bDerivByte12[1] = r[43] & r[44] & r[52] ^ r[43] & r[44] ^ r[43] & r[51] ^ r[43] & r[52] ^ r[44] & r[51] & r[52] ^ r[44] & r[51] ^ r[44] ^ r[51] & r[52] ^ r[51] ^ r[52];
                               bDerivByte12[0] = r[44] & r[45] & r[53] ^ r[44] & r[52] & r[53] ^ r[44] & r[52] ^ r[45] & r[52] & r[53] ^ r[45] ^ r[52] ^ r[53] ^ 0xff;*/ //old

                bDerivByte12[7] = r[50] & r[49] & r[42] ^ r[50] & r[49] ^ r[50] & r[42] & r[41] ^ r[50] & r[42] ^ r[50] & r[41] ^ r[49] & r[42] & r[41] ^ r[42] ^ r[41] ^ 0xff;
                bDerivByte12[6] = r[49] & r[48] & r[41] ^ r[49] & r[48] ^ r[49] & r[41] & r[40] ^ r[49] & r[40] ^ r[49] ^ r[48] & r[40] ^ r[48] ^ r[41] ^ 0xff;
                bDerivByte12[5] = r[55] & r[48] & r[47] ^ r[55] & r[48] & r[40] ^ r[55] & r[47] & r[40] ^ r[55] & r[47] ^ r[55] ^ r[48] & r[47] & r[40] ^ r[47];
                bDerivByte12[4] = r[55] & r[54] & r[47] ^ r[55] & r[54] & r[46] ^ r[55] & r[47] & r[46] ^ r[54] & r[47] & r[46] ^ r[54] & r[47] ^ r[54] ^ r[47] & r[46] ^ r[46];
                bDerivByte12[3] = r[54] & r[53] & r[46] ^ r[54] & r[53] & r[45] ^ r[54] & r[53] ^ r[54] & r[46] & r[45] ^ r[54] & r[46] ^ r[54] ^ r[53] & r[46] & r[45] ^ r[53] ^ r[46] & r[45] ^ r[45];
                bDerivByte12[2] = r[53] & r[52] & r[44] ^ r[53] & r[45] ^ r[52] & r[45] & r[44] ^ r[52] & r[44] ^ r[52] ^ r[44];
                bDerivByte12[1] = r[52] & r[51] & r[43] ^ r[52] & r[51] ^ r[52] & r[44] ^ r[52] & r[43] ^ r[51] & r[44] & r[43] ^ r[51] & r[44] ^ r[51] ^ r[44] & r[43] ^ r[44] ^ r[43];
                bDerivByte12[0] = r[51] & r[50] & r[42] ^ r[51] & r[43] & r[42] ^ r[51] & r[43] ^ r[50] & r[43] & r[42] ^ r[50] ^ r[43] ^ r[42] ^ 0xff;

                //                uint8_t bDerivByte12 = gpu_tea3_state_word_to_newbyte((qwIvReg >> 8) & 0xffff, gpu_awTea3LutA);  // F32
                //                uint8_t bDerivByte56 = gpu_tea3_state_word_to_newbyte((qwIvReg >> 40) & 0xffff, gpu_awTea3LutB); // F31
                /*                bDerivByte56[7] = r[13] & r[14] & r[21] ^ r[13] & r[14] ^ r[13] & r[21] & r[22] ^ r[13] & r[21] ^ r[13] & r[22] ^ r[14] & r[21] & r[22] ^ r[21] & r[22] ^ r[21] ^ r[22];
                                bDerivByte56[6] = r[14] & r[15] & r[22] ^ r[14] & r[15] ^ r[14] & r[22] & r[23] ^ r[14] & r[23] ^ r[14] ^ r[15] & r[22] ^ r[15] & r[23] ^ r[22] & r[23] ^ r[22] ^ r[23] ^ 0xff;
                                bDerivByte56[5] = r[8] & r[15] & r[16] ^ r[8] & r[16] & r[23] ^ r[8] & r[16] ^ r[8] ^ r[15] & r[23] ^ r[16];
                                bDerivByte56[4] = r[8] & r[9] & r[16] ^ r[8] & r[9] & r[17] ^ r[8] & r[9] ^ r[8] & r[16] & r[17] ^ r[8] & r[17] ^ r[9] & r[16] & r[17] ^ r[9] ^ r[17];
                                bDerivByte56[3] = r[9] & r[10] & r[18] ^ r[9] & r[10] ^ r[9] & r[17] ^ r[9] & r[18] ^ r[10] & r[17] & r[18] ^ r[10] & r[17] ^ r[10] ^ r[17] & r[18] ^ r[17] ^ r[18] ^ 0xff;
                                bDerivByte56[2] = r[10] & r[11] & r[19] ^ r[10] & r[18] ^ r[10] ^ r[11] & r[18] & r[19] ^ r[11] & r[19] ^ r[11] ^ r[18] ^ r[19] ^ 0xff;
                                bDerivByte56[1] = r[11] & r[12] & r[20] ^ r[11] & r[19] ^ r[11] & r[20] ^ r[12] & r[19] & r[20] ^ r[12] & r[20] ^ r[12] ^ r[19] & r[20] ^ 0xff;
                                bDerivByte56[0] = r[12] & r[13] & r[21] ^ r[12] & r[20] & r[21] ^ r[12] & r[20] ^ r[12] & r[21] ^ r[13] & r[20] & r[21] ^ r[13] ^ r[21] ^ 0xff;*/ //old


                bDerivByte56[7] = r[18] & r[17] & r[10] ^ r[18] & r[17] ^ r[18] & r[10] & r[9] ^ r[18] & r[10] ^ r[18] & r[9] ^ r[17] & r[10] & r[9] ^ r[10] & r[9] ^ r[10] ^ r[9];
                bDerivByte56[6] = r[17] & r[16] & r[9] ^ r[17] & r[16] ^ r[17] & r[9] & r[8] ^ r[17] & r[8] ^ r[17] ^ r[16] & r[9] ^ r[16] & r[8] ^ r[9] & r[8] ^ r[9] ^ r[8] ^ 0xff;
                bDerivByte56[5] = r[23] & r[16] & r[15] ^ r[23] & r[15] & r[8] ^ r[23] & r[15] ^ r[23] ^ r[16] & r[8] ^ r[15];
                bDerivByte56[4] = r[23] & r[22] & r[15] ^ r[23] & r[22] & r[14] ^ r[23] & r[22] ^ r[23] & r[15] & r[14] ^ r[23] & r[14] ^ r[22] & r[15] & r[14] ^ r[22] ^ r[14];
                bDerivByte56[3] = r[22] & r[21] & r[13] ^ r[22] & r[21] ^ r[22] & r[14] ^ r[22] & r[13] ^ r[21] & r[14] & r[13] ^ r[21] & r[14] ^ r[21] ^ r[14] & r[13] ^ r[14] ^ r[13] ^ 0xff;
                bDerivByte56[2] = r[21] & r[20] & r[12] ^ r[21] & r[13] ^ r[21] ^ r[20] & r[13] & r[12] ^ r[20] & r[12] ^ r[20] ^ r[13] ^ r[12] ^ 0xff;
                bDerivByte56[1] = r[20] & r[19] & r[11] ^ r[20] & r[12] ^ r[20] & r[11] ^ r[19] & r[12] & r[11] ^ r[19] & r[11] ^ r[19] ^ r[12] & r[11] ^ 0xff;
                bDerivByte56[0] = r[19] & r[18] & r[10] ^ r[19] & r[11] & r[10] ^ r[19] & r[11] ^ r[19] & r[10] ^ r[18] & r[11] & r[10] ^ r[18] ^ r[10] ^ 0xff;


                //                uint8_t bReordByte4 = gpu_tea3_reorder_state_byte((qwIvReg >> 32) & 0xff);
                bReordByte4[0] = r[26];
                bReordByte4[1] = r[31];
                bReordByte4[2] = r[27];
                bReordByte4[3] = r[29];
                bReordByte4[4] = r[30];
                bReordByte4[5] = r[25];
                bReordByte4[6] = r[24];
                bReordByte4[7] = r[28];
                //               if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bDerivByte12[i]); printf("\n");}
                 //              if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bDerivByte56[i]); printf("\n"); }
                 //              if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bReordByte4[i]); printf("\n"); }
                               // Transpose the S-box output
                int k;
                unsigned m, t;
                m = 0x0000000F;

                for (int l = 4; l != 0; l = l >> 1, m = m ^ (m << l)) {
                    for (k = 0; k < 8; k = (k + l + 1) & ~l) {
                        t = (bSboxOut[k] ^ (bSboxOut[k + l] >> l)) & m;
                        bSboxOut[k] = bSboxOut[k] ^ t;
                        bSboxOut[k + l] = bSboxOut[k + l] ^ (t << l);
                    }
                }

                // Transpose the S-box (my implementation)
 /*               uint8_t bSboxOut2[8] = { 0 };
                for (int i = 0; i < 8; i++) {
                    for (int j = 7; j > 0; j--) {
                        bSboxOut2[i] |= bSboxOut[j] & (0x80 >> i);
                        bSboxOut2[i] >>= 1;
                    }
                    bSboxOut2[i] |= bSboxOut[0] & (0x80 >> i);
                }*/

                //               if (threadIndex == 0) for (int i = 0; i < 8; i++) printf("%02x\n", bSboxOut[i]); // Sboxes are transposed correctly
                               // Step 3: Combine current state with state derived values, and xor in key derived sbox output
               //                uint8_t bNewByte = ((qwIvReg >> 56) ^ bReordByte4 ^ bDerivByte12 ^ bSboxOut) & 0xff;
               /*                bNewByte[0] = r[0] ^ bReordByte4[0] ^ bDerivByte12[0] ^ bSboxOut[0]; // bSboxOut must be turned into bSboxOut[8]
                               bNewByte[1] = r[1] ^ bReordByte4[1] ^ bDerivByte12[1] ^ bSboxOut[1]; // bSboxOut must be turned into bSboxOut[8]
                               bNewByte[2] = r[2] ^ bReordByte4[2] ^ bDerivByte12[2] ^ bSboxOut[2]; // bSboxOut must be turned into bSboxOut[8]
                               bNewByte[3] = r[3] ^ bReordByte4[3] ^ bDerivByte12[3] ^ bSboxOut[3]; // bSboxOut must be turned into bSboxOut[8]
                               bNewByte[4] = r[4] ^ bReordByte4[4] ^ bDerivByte12[4] ^ bSboxOut[4]; // bSboxOut must be turned into bSboxOut[8]
                               bNewByte[5] = r[5] ^ bReordByte4[5] ^ bDerivByte12[5] ^ bSboxOut[5]; // bSboxOut must be turned into bSboxOut[8]
                               bNewByte[6] = r[6] ^ bReordByte4[6] ^ bDerivByte12[6] ^ bSboxOut[6]; // bSboxOut must be turned into bSboxOut[8]
                               bNewByte[7] = r[7] ^ bReordByte4[7] ^ bDerivByte12[7] ^ bSboxOut[7]; // bSboxOut must be turned into bSboxOut[8]
               */
                for (int i = 0; i < 8; i++) bNewByte[i] = r[i] ^ bReordByte4[i] ^ bDerivByte12[i] ^ bSboxOut[i];



                //               uint8_t bMixByte = bDerivByte56;
                               // Step 4: Update lfsr: leftshift 8, feed/mix in previously generated bytes
                //               qwIvReg = ((qwIvReg << 8) ^ ((uint64_t)bMixByte << 40)) | bNewByte;

                r[0] = r[8];
                r[1] = r[9];
                r[2] = r[10];
                r[3] = r[11];
                r[4] = r[12];
                r[5] = r[13];
                r[6] = r[14];
                r[7] = r[15];
                r[8] = r[16];
                r[9] = r[17];
                r[10] = r[18];
                r[11] = r[19];
                r[12] = r[20];
                r[13] = r[21];
                r[14] = r[22];
                r[15] = r[23];
                r[16] = r[24] ^ bDerivByte56[0];
                r[17] = r[25] ^ bDerivByte56[1];
                r[18] = r[26] ^ bDerivByte56[2];
                r[19] = r[27] ^ bDerivByte56[3];
                r[20] = r[28] ^ bDerivByte56[4];
                r[21] = r[29] ^ bDerivByte56[5];
                r[22] = r[30] ^ bDerivByte56[6];
                r[23] = r[31] ^ bDerivByte56[7];
                r[24] = r[32];
                r[25] = r[33];
                r[26] = r[34];
                r[27] = r[35];
                r[28] = r[36];
                r[29] = r[37];
                r[30] = r[38];
                r[31] = r[39];
                r[32] = r[40];
                r[33] = r[41];
                r[34] = r[42];
                r[35] = r[43];
                r[36] = r[44];
                r[37] = r[45];
                r[38] = r[46];
                r[39] = r[47];
                r[40] = r[48];
                r[41] = r[49];
                r[42] = r[50];
                r[43] = r[51];
                r[44] = r[52];
                r[45] = r[53];
                r[46] = r[54];
                r[47] = r[55];
                r[48] = r[56];
                r[49] = r[57];
                r[50] = r[58];
                r[51] = r[59];
                r[52] = r[60];
                r[53] = r[61];
                r[54] = r[62];
                r[55] = r[63];

                r[56] = bNewByte[0];
                r[57] = bNewByte[1];
                r[58] = bNewByte[2];
                r[59] = bNewByte[3];
                r[60] = bNewByte[4];
                r[61] = bNewByte[5];
                r[62] = bNewByte[6];
                r[63] = bNewByte[7];

            }
            //           lpKsOut[i] = (qwIvReg >> 56);
 //           if ((qwIvReg >> 56) != lpKsOut[i]) flag = 0;
            for (int i = 0; i < 8; i++)
                for (int c = 0; c < 8; c++)
                    if ((r[c] & (0x80 >> i)) != (lpKsOut[w * 8 + c] & (0x80 >> i))) flag[i] = 0;
            //            if (threadIndex == 0 && trial==0) printf("%llx %x\n", qwIvReg >> 56, lpKsOut[i]);
            dwNumSkipRounds = 19;
            flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7];
        }
        //        if (threadIndex == 0) for (int i = 0; i < 8;i++) printf("%x\n", r[i]);
        if (flag_overall == 1) { gpu_captured_key[0] = threadIndex; gpu_captured_key[1] = trial; }
        //        if (flag == 1) printf("Hello world %llu\n",threadIndex);

    }
}
// tea3_exhaustive_bitsliced16 kernel keeps each bit of the state in a 16-bit variable
// These 16-bit variables keep 16 different states so one encryption performed by a thread actually means 16 encryptions in parallel
__global__ void tea3_exhaustive_bitsliced16(uint16_t* reg, uint32_t dwNumKsBytes, uint16_t* lpKsOut_d, uint8_t* gpu_abTea3Sbox_d, int trials, uint64_t* gpu_captured_key) {
    __shared__ uint8_t gpu_abTea3Sbox[256];
    __shared__ uint16_t lpKsOut[80];
    if (threadIdx.x < 256) {
        gpu_abTea3Sbox[threadIdx.x] = gpu_abTea3Sbox_d[threadIdx.x];
        if (threadIdx.x < 80) lpKsOut[threadIdx.x] = lpKsOut_d[threadIdx.x];
    }
    __syncthreads();
    uint16_t key_right[16]; // rightmost 16 bits
    uint64_t key_left[16]; // leftmost 64 bits
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint16_t r[64];
    uint16_t bDerivByte12[8], bDerivByte56[8], bReordByte4[8], bNewByte[8];
    int flag[16];
    for (int trial = 0; trial < trials; trial++) {
        uint32_t dwNumSkipRounds = 51;
        for (int i = 0; i < 64; i++) r[i] = reg[i];
        for (uint64_t i = 0; i < 16; i++) {
            key_right[i] = trial; // rightmost 16 bits
            key_left[i] = (threadIndex << (64 - BLOCKSLOG - THREADSLOG)) ^ (i << 16); // leftmost 64 bits
            flag[i] = 1;
        }
        int flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7] | flag[8] | flag[9] | flag[10] | flag[11] | flag[12] | flag[13] | flag[14] | flag[15];
        for (int w = 0; (w < dwNumKsBytes) && flag_overall == 1; w++) {
            for (int j = 0; j < dwNumSkipRounds; j++) {
                // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
                uint16_t bSboxOut[16] = { 0 };
                for (int i = 0; i < 16; i++) {
                    bSboxOut[i] = gpu_abTea3Sbox[(key_left[i] & 0xff) ^ ((key_left[i] >> 40) & 0xff)] ^ (key_left[i] >> 56);
                    key_left[i] = (key_left[i] << 8) ^ (key_right[i] >> 8);
                    key_right[i] = (key_right[i] << 8) ^ (bSboxOut[i]&0xff);
                }
                //              if (threadIndex == 0) for (int i = 0; i < 16; i++) printf("%llx %x\n",key_left[i], key_right[i]); // keys are generated correctly 
                //              if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 64; i++) printf("%02x\n", reg[i]); printf("\n"); }
                bDerivByte12[7] = r[50] & r[49] & r[42] ^ r[50] & r[49] ^ r[50] & r[42] & r[41] ^ r[50] & r[42] ^ r[50] & r[41] ^ r[49] & r[42] & r[41] ^ r[42] ^ r[41] ^ 0xffff;
                bDerivByte12[6] = r[49] & r[48] & r[41] ^ r[49] & r[48] ^ r[49] & r[41] & r[40] ^ r[49] & r[40] ^ r[49] ^ r[48] & r[40] ^ r[48] ^ r[41] ^ 0xffff;
                bDerivByte12[5] = r[55] & r[48] & r[47] ^ r[55] & r[48] & r[40] ^ r[55] & r[47] & r[40] ^ r[55] & r[47] ^ r[55] ^ r[48] & r[47] & r[40] ^ r[47];
                bDerivByte12[4] = r[55] & r[54] & r[47] ^ r[55] & r[54] & r[46] ^ r[55] & r[47] & r[46] ^ r[54] & r[47] & r[46] ^ r[54] & r[47] ^ r[54] ^ r[47] & r[46] ^ r[46];
                bDerivByte12[3] = r[54] & r[53] & r[46] ^ r[54] & r[53] & r[45] ^ r[54] & r[53] ^ r[54] & r[46] & r[45] ^ r[54] & r[46] ^ r[54] ^ r[53] & r[46] & r[45] ^ r[53] ^ r[46] & r[45] ^ r[45];
                bDerivByte12[2] = r[53] & r[52] & r[44] ^ r[53] & r[45] ^ r[52] & r[45] & r[44] ^ r[52] & r[44] ^ r[52] ^ r[44];
                bDerivByte12[1] = r[52] & r[51] & r[43] ^ r[52] & r[51] ^ r[52] & r[44] ^ r[52] & r[43] ^ r[51] & r[44] & r[43] ^ r[51] & r[44] ^ r[51] ^ r[44] & r[43] ^ r[44] ^ r[43];
                bDerivByte12[0] = r[51] & r[50] & r[42] ^ r[51] & r[43] & r[42] ^ r[51] & r[43] ^ r[50] & r[43] & r[42] ^ r[50] ^ r[43] ^ r[42] ^ 0xffff;

                bDerivByte56[7] = r[18] & r[17] & r[10] ^ r[18] & r[17] ^ r[18] & r[10] & r[9] ^ r[18] & r[10] ^ r[18] & r[9] ^ r[17] & r[10] & r[9] ^ r[10] & r[9] ^ r[10] ^ r[9];
                bDerivByte56[6] = r[17] & r[16] & r[9] ^ r[17] & r[16] ^ r[17] & r[9] & r[8] ^ r[17] & r[8] ^ r[17] ^ r[16] & r[9] ^ r[16] & r[8] ^ r[9] & r[8] ^ r[9] ^ r[8] ^ 0xffff;
                bDerivByte56[5] = r[23] & r[16] & r[15] ^ r[23] & r[15] & r[8] ^ r[23] & r[15] ^ r[23] ^ r[16] & r[8] ^ r[15];
                bDerivByte56[4] = r[23] & r[22] & r[15] ^ r[23] & r[22] & r[14] ^ r[23] & r[22] ^ r[23] & r[15] & r[14] ^ r[23] & r[14] ^ r[22] & r[15] & r[14] ^ r[22] ^ r[14];
                bDerivByte56[3] = r[22] & r[21] & r[13] ^ r[22] & r[21] ^ r[22] & r[14] ^ r[22] & r[13] ^ r[21] & r[14] & r[13] ^ r[21] & r[14] ^ r[21] ^ r[14] & r[13] ^ r[14] ^ r[13] ^ 0xffff;
                bDerivByte56[2] = r[21] & r[20] & r[12] ^ r[21] & r[13] ^ r[21] ^ r[20] & r[13] & r[12] ^ r[20] & r[12] ^ r[20] ^ r[13] ^ r[12] ^ 0xffff;
                bDerivByte56[1] = r[20] & r[19] & r[11] ^ r[20] & r[12] ^ r[20] & r[11] ^ r[19] & r[12] & r[11] ^ r[19] & r[11] ^ r[19] ^ r[12] & r[11] ^ 0xffff;
                bDerivByte56[0] = r[19] & r[18] & r[10] ^ r[19] & r[11] & r[10] ^ r[19] & r[11] ^ r[19] & r[10] ^ r[18] & r[11] & r[10] ^ r[18] ^ r[10] ^ 0xffff;

                bReordByte4[0] = r[26];
                bReordByte4[1] = r[31];
                bReordByte4[2] = r[27];
                bReordByte4[3] = r[29];
                bReordByte4[4] = r[30];
                bReordByte4[5] = r[25];
                bReordByte4[6] = r[24];
                bReordByte4[7] = r[28];

//                              if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bDerivByte12[i]); printf("\n");}
//              if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bDerivByte56[i]); printf("\n"); }
//               if (threadIndex == 0) { printf("\n"); for (int i = 0; i < 8; i++) printf("%x\n", bReordByte4[i]); printf("\n"); }
                               // Transpose the S-box output
                int k;
                unsigned m, t;
                m = 0x000000FF;

                for (int l = 8; l != 0; l = l >> 1, m = m ^ (m << l)) {
                    for (k = 0; k < 16; k = (k + l + 1) & ~l) {
                        t = (bSboxOut[k] ^ (bSboxOut[k + l] >> l)) & m;
                        bSboxOut[k] = bSboxOut[k] ^ t;
                        bSboxOut[k + l] = bSboxOut[k + l] ^ (t << l);
                    }
                }

                for (int i = 0; i < 8; i++) bNewByte[i] = r[i] ^ bReordByte4[i] ^ bDerivByte12[i] ^ bSboxOut[i+8];

                r[0] = r[8];
                r[1] = r[9];
                r[2] = r[10];
                r[3] = r[11];
                r[4] = r[12];
                r[5] = r[13];
                r[6] = r[14];
                r[7] = r[15];
                r[8] = r[16];
                r[9] = r[17];
                r[10] = r[18];
                r[11] = r[19];
                r[12] = r[20];
                r[13] = r[21];
                r[14] = r[22];
                r[15] = r[23];
                r[16] = r[24] ^ bDerivByte56[0];
                r[17] = r[25] ^ bDerivByte56[1];
                r[18] = r[26] ^ bDerivByte56[2];
                r[19] = r[27] ^ bDerivByte56[3];
                r[20] = r[28] ^ bDerivByte56[4];
                r[21] = r[29] ^ bDerivByte56[5];
                r[22] = r[30] ^ bDerivByte56[6];
                r[23] = r[31] ^ bDerivByte56[7];
                r[24] = r[32];
                r[25] = r[33];
                r[26] = r[34];
                r[27] = r[35];
                r[28] = r[36];
                r[29] = r[37];
                r[30] = r[38];
                r[31] = r[39];
                r[32] = r[40];
                r[33] = r[41];
                r[34] = r[42];
                r[35] = r[43];
                r[36] = r[44];
                r[37] = r[45];
                r[38] = r[46];
                r[39] = r[47];
                r[40] = r[48];
                r[41] = r[49];
                r[42] = r[50];
                r[43] = r[51];
                r[44] = r[52];
                r[45] = r[53];
                r[46] = r[54];
                r[47] = r[55];
                r[48] = r[56];
                r[49] = r[57];
                r[50] = r[58];
                r[51] = r[59];
                r[52] = r[60];
                r[53] = r[61];
                r[54] = r[62];
                r[55] = r[63];

                r[56] = bNewByte[0];
                r[57] = bNewByte[1];
                r[58] = bNewByte[2];
                r[59] = bNewByte[3];
                r[60] = bNewByte[4];
                r[61] = bNewByte[5];
                r[62] = bNewByte[6];
                r[63] = bNewByte[7];

            }
            for (int i = 0; i < 16; i++)
                for (int c = 0; c < 8; c++)
                    if ((r[c] & (0x8000 >> i)) != (lpKsOut[w * 8 + c] & (0x8000 >> i))) flag[i] = 0;
            dwNumSkipRounds = 19;
            flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7] | flag[8] | flag[9] | flag[10] | flag[11] | flag[12] | flag[13] | flag[14] | flag[15];
        }
        if (flag_overall == 1) { gpu_captured_key[0] = threadIndex; gpu_captured_key[1] = trial; }
    }
}
// tea3_exhaustive_bitsliced32 kernel keeps each bit of the state in a 32-bit variable
// These 32-bit variables keep 32 different states so one encryption performed by a thread actually means 32 encryptions in parallel
__global__ void tea3_exhaustive_bitsliced32(uint32_t* reg, uint32_t dwNumKsBytes, uint32_t* lpKsOut_d, uint8_t* gpu_abTea3Sbox_d, int trials, uint64_t* gpu_captured_key) {
    __shared__ uint8_t gpu_abTea3Sbox[256];
    __shared__ uint32_t lpKsOut[80];
    if (threadIdx.x < 256) {
        gpu_abTea3Sbox[threadIdx.x] = gpu_abTea3Sbox_d[threadIdx.x];
        if (threadIdx.x < 80) lpKsOut[threadIdx.x] = lpKsOut_d[threadIdx.x];
    }
    __syncthreads();
    uint16_t key_right[16]; // rightmost 16 bits
    uint64_t key_left[16]; // leftmost 64 bits
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint32_t r[64];
    uint32_t bDerivByte12[8], bDerivByte56[8], bReordByte4[8], bNewByte[8];
    int flag[32];
    for (int trial = 0; trial < trials; trial++) {
        uint32_t dwNumSkipRounds = 51;
#pragma unroll
        for (int i = 0; i < 64; i++) r[i] = reg[i];
#pragma unroll
        for (uint64_t i = 0; i < 32; i++) {
            key_right[i] = trial; // rightmost 16 bits
            key_left[i] = threadIndex ^ (i << 36); // leftmost 64 bits
            flag[i] = 1;
        }
        int flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7] | flag[8] | flag[9] | flag[10] | flag[11] | flag[12] | flag[13] | flag[14] | flag[15]
            | flag[16] | flag[17] | flag[18] | flag[19] | flag[20] | flag[21] | flag[22] | flag[23] | flag[24] | flag[25] | flag[26] | flag[27] | flag[28] | flag[29] | flag[30] | flag[31];
        for (int w = 0; (w < dwNumKsBytes) && flag_overall == 1; w++) {
            for (int j = 0; j < dwNumSkipRounds; j++) {
                // Step 1: Derive a non-linear feedback byte through sbox and feed back into key register
                uint32_t bSboxOut[32] = { 0 };
#pragma unroll
                for (int i = 0; i < 32; i++) {
                    bSboxOut[i] = gpu_abTea3Sbox[(key_left[i] & 0xff) ^ ((key_left[i] >> 40) & 0xff)] ^ (key_left[i] >> 56);
                    key_left[i] = (key_left[i] << 8) ^ (key_right[i] >> 8);
                    key_right[i] = (key_right[i] << 8) ^ (bSboxOut[i] & 0xff);
                }
                bDerivByte12[7] = r[50] & r[49] & r[42] ^ r[50] & r[49] ^ r[50] & r[42] & r[41] ^ r[50] & r[42] ^ r[50] & r[41] ^ r[49] & r[42] & r[41] ^ r[42] ^ r[41] ^ 0xffffffff;
                bDerivByte12[6] = r[49] & r[48] & r[41] ^ r[49] & r[48] ^ r[49] & r[41] & r[40] ^ r[49] & r[40] ^ r[49] ^ r[48] & r[40] ^ r[48] ^ r[41] ^ 0xffffffff;
                bDerivByte12[5] = r[55] & r[48] & r[47] ^ r[55] & r[48] & r[40] ^ r[55] & r[47] & r[40] ^ r[55] & r[47] ^ r[55] ^ r[48] & r[47] & r[40] ^ r[47];
                bDerivByte12[4] = r[55] & r[54] & r[47] ^ r[55] & r[54] & r[46] ^ r[55] & r[47] & r[46] ^ r[54] & r[47] & r[46] ^ r[54] & r[47] ^ r[54] ^ r[47] & r[46] ^ r[46];
                bDerivByte12[3] = r[54] & r[53] & r[46] ^ r[54] & r[53] & r[45] ^ r[54] & r[53] ^ r[54] & r[46] & r[45] ^ r[54] & r[46] ^ r[54] ^ r[53] & r[46] & r[45] ^ r[53] ^ r[46] & r[45] ^ r[45];
                bDerivByte12[2] = r[53] & r[52] & r[44] ^ r[53] & r[45] ^ r[52] & r[45] & r[44] ^ r[52] & r[44] ^ r[52] ^ r[44];
                bDerivByte12[1] = r[52] & r[51] & r[43] ^ r[52] & r[51] ^ r[52] & r[44] ^ r[52] & r[43] ^ r[51] & r[44] & r[43] ^ r[51] & r[44] ^ r[51] ^ r[44] & r[43] ^ r[44] ^ r[43];
                bDerivByte12[0] = r[51] & r[50] & r[42] ^ r[51] & r[43] & r[42] ^ r[51] & r[43] ^ r[50] & r[43] & r[42] ^ r[50] ^ r[43] ^ r[42] ^ 0xffffffff;

                bDerivByte56[7] = r[18] & r[17] & r[10] ^ r[18] & r[17] ^ r[18] & r[10] & r[9] ^ r[18] & r[10] ^ r[18] & r[9] ^ r[17] & r[10] & r[9] ^ r[10] & r[9] ^ r[10] ^ r[9];
                bDerivByte56[6] = r[17] & r[16] & r[9] ^ r[17] & r[16] ^ r[17] & r[9] & r[8] ^ r[17] & r[8] ^ r[17] ^ r[16] & r[9] ^ r[16] & r[8] ^ r[9] & r[8] ^ r[9] ^ r[8] ^ 0xffffffff;
                bDerivByte56[5] = r[23] & r[16] & r[15] ^ r[23] & r[15] & r[8] ^ r[23] & r[15] ^ r[23] ^ r[16] & r[8] ^ r[15];
                bDerivByte56[4] = r[23] & r[22] & r[15] ^ r[23] & r[22] & r[14] ^ r[23] & r[22] ^ r[23] & r[15] & r[14] ^ r[23] & r[14] ^ r[22] & r[15] & r[14] ^ r[22] ^ r[14];
                bDerivByte56[3] = r[22] & r[21] & r[13] ^ r[22] & r[21] ^ r[22] & r[14] ^ r[22] & r[13] ^ r[21] & r[14] & r[13] ^ r[21] & r[14] ^ r[21] ^ r[14] & r[13] ^ r[14] ^ r[13] ^ 0xffffffff;
                bDerivByte56[2] = r[21] & r[20] & r[12] ^ r[21] & r[13] ^ r[21] ^ r[20] & r[13] & r[12] ^ r[20] & r[12] ^ r[20] ^ r[13] ^ r[12] ^ 0xffffffff;
                bDerivByte56[1] = r[20] & r[19] & r[11] ^ r[20] & r[12] ^ r[20] & r[11] ^ r[19] & r[12] & r[11] ^ r[19] & r[11] ^ r[19] ^ r[12] & r[11] ^ 0xffffffff;
                bDerivByte56[0] = r[19] & r[18] & r[10] ^ r[19] & r[11] & r[10] ^ r[19] & r[11] ^ r[19] & r[10] ^ r[18] & r[11] & r[10] ^ r[18] ^ r[10] ^ 0xffffffff;

                bReordByte4[0] = r[26];
                bReordByte4[1] = r[31];
                bReordByte4[2] = r[27];
                bReordByte4[3] = r[29];
                bReordByte4[4] = r[30];
                bReordByte4[5] = r[25];
                bReordByte4[6] = r[24];
                bReordByte4[7] = r[28];

                // Transpose the S-box output
                int k;
                unsigned m, t;
                m = 0x0000FFFF;
#pragma unroll
                for (int l = 16; l != 0; l = l >> 1, m = m ^ (m << l)) {
#pragma unroll
                    for (k = 0; k < 32; k = (k + l + 1) & ~l) {
                        t = (bSboxOut[k] ^ (bSboxOut[k + l] >> l)) & m;
                        bSboxOut[k] = bSboxOut[k] ^ t;
                        bSboxOut[k + l] = bSboxOut[k + l] ^ (t << l);
                    }
                }
#pragma unroll
                for (int i = 0; i < 8; i++) bNewByte[i] = r[i] ^ bReordByte4[i] ^ bDerivByte12[i] ^ bSboxOut[i + 24];
                r[0] = r[8];
                r[1] = r[9];
                r[2] = r[10];
                r[3] = r[11];
                r[4] = r[12];
                r[5] = r[13];
                r[6] = r[14];
                r[7] = r[15];
                r[8] = r[16];
                r[9] = r[17];
                r[10] = r[18];
                r[11] = r[19];
                r[12] = r[20];
                r[13] = r[21];
                r[14] = r[22];
                r[15] = r[23];
                r[16] = r[24] ^ bDerivByte56[0];
                r[17] = r[25] ^ bDerivByte56[1];
                r[18] = r[26] ^ bDerivByte56[2];
                r[19] = r[27] ^ bDerivByte56[3];
                r[20] = r[28] ^ bDerivByte56[4];
                r[21] = r[29] ^ bDerivByte56[5];
                r[22] = r[30] ^ bDerivByte56[6];
                r[23] = r[31] ^ bDerivByte56[7];
                r[24] = r[32];
                r[25] = r[33];
                r[26] = r[34];
                r[27] = r[35];
                r[28] = r[36];
                r[29] = r[37];
                r[30] = r[38];
                r[31] = r[39];
                r[32] = r[40];
                r[33] = r[41];
                r[34] = r[42];
                r[35] = r[43];
                r[36] = r[44];
                r[37] = r[45];
                r[38] = r[46];
                r[39] = r[47];
                r[40] = r[48];
                r[41] = r[49];
                r[42] = r[50];
                r[43] = r[51];
                r[44] = r[52];
                r[45] = r[53];
                r[46] = r[54];
                r[47] = r[55];
                r[48] = r[56];
                r[49] = r[57];
                r[50] = r[58];
                r[51] = r[59];
                r[52] = r[60];
                r[53] = r[61];
                r[54] = r[62];
                r[55] = r[63];

                r[56] = bNewByte[0];
                r[57] = bNewByte[1];
                r[58] = bNewByte[2];
                r[59] = bNewByte[3];
                r[60] = bNewByte[4];
                r[61] = bNewByte[5];
                r[62] = bNewByte[6];
                r[63] = bNewByte[7];

            }
#pragma unroll
            for (int i = 0; i < 32; i++)
#pragma unroll
                for (int c = 0; c < 8; c++)
                    if ((r[c] & (0x80000000 >> i)) != (lpKsOut[w * 8 + c] & (0x80000000 >> i))) flag[i] = 0;
            dwNumSkipRounds = 19;
            flag_overall = flag[0] | flag[1] | flag[2] | flag[3] | flag[4] | flag[5] | flag[6] | flag[7] | flag[8] | flag[9] | flag[10] | flag[11] | flag[12] | flag[13] | flag[14] | flag[15]
                | flag[16] | flag[17] | flag[18] | flag[19] | flag[20] | flag[21] | flag[22] | flag[23] | flag[24] | flag[25] | flag[26] | flag[27] | flag[28] | flag[29] | flag[30] | flag[31];
        }
        if (flag_overall == 1) { gpu_captured_key[0] = threadIndex; gpu_captured_key[1] = trial; }
    }
}

void user_input() {
    // Default values of BLOCKS=1024, THREADS=256 mean that a kernel creates 2^18 threads.
    // How many encryptions each thread is going to perform is determined by the user as a power of 2
    // Thus, for a non-bitsliced implementation, a user input of 5 means 2^23 encryptions for a kernel
    // For a bitsliced implementation where 32-bit values store 32 different states, 2^18 threads performe 2^23 encryptions
    
    printf("(1) Exhaustive search on 80-bit keystream\n"
        "(2) Exhaustive key search on 80-bit keystream (0 shared memory bank conflicts)\n"
        "(3) Exhaustive key search on 80-bit keystream (some shared memory bank conflicts)\n"
        "(4) Keystream generation of 80 bits\n"
        "(5) Keystream generation of 80 bits (0 shared memory bank conflicts)\n"
        "(6) Keystream generation of 80 bits(some shared memory bank conflicts)\n"
        "...\n"
        "(11) BITSLICED Exhaustive search on 80-bit keystream (Each thread performs 8 encrypitons in parallel)\n"
        "(12) BITSLICED Exhaustive search on 80-bit keystream (Each thread performs 8 encrypitons in parallel) (0 bank conflicts)\n"
        "(13) BITSLICED Exhaustive search on 80-bit keystream (Each thread performs 16 encrypitons in parallel)\n"
        "(14) BITSLICED Exhaustive search on 80-bit keystream (Each thread performs 32 encrypitons in parallel)\n"
        "Choice: "
    );
    scanf_s("%d", &choice);
    if (choice == 11) printf("Trials 2^21 + ");
    else if (choice == 12) printf("Trials 2^21 + ");
    else if (choice == 13) printf("Trials 2^22 + ");
    else if (choice == 14) printf("Trials 2^23 + ");
    else printf("Trials 2^18 + ");
    scanf_s("%d", &trials);
    trials = 1 << trials;
}

int main() {
    hipSetDevice(0);
    user_input();
    uint16_t *gpu_awTea3LutA;
    uint16_t *gpu_awTea3LutB;
    uint8_t *gpu_abTea3Sbox;
    uint8_t *gpu_lpKsOut;
    uint64_t *gpu_captured_key; uint64_t captured_key[2] = { 0xffffffffffffffff, 0xffffffffffffffff };    
    uint8_t bitsliced_keystream[80] = { 0 }; uint16_t bitsliced_keystream16[80] = { 0 }; uint32_t bitsliced_keystream32[80] = { 0 };
    uint8_t *bitsliced_keystream_d; uint16_t* bitsliced_keystream16_d; uint32_t* bitsliced_keystream32_d;
    uint32_t dwNumKsBytes = 10;
    // Test vectors 1
 //   uint32_t dwFrameNumbers = 0;// 0xffffffff;
 //   uint8_t lpKsOut[10] = { 0x3b, 0x35, 0x44, 0x30, 0xdc, 0x3d, 0x3f, 0xee, 0x76, 0xcf };
    // Test vectors 2
    uint32_t dwFrameNumbers = 0x176C;
    uint8_t lpKsOut[10] = { 0x88, 0x40, 0x58, 0x9f, 0x88, 0x7b, 0x93, 0xa5, 0xac, 0x91 };
    uint64_t qwIvReg = tea3_compute_iv(dwFrameNumbers);
    uint8_t reg[64] = {0};    uint16_t reg16[64] = { 0 }; uint32_t reg32[64] = { 0 };
    uint8_t *reg_d; uint16_t* reg16_d; uint32_t* reg32_d;
    for (int i = 0; i < 64; i++)
        for (int j = 0; j < 8; j++) {
            reg[i] = reg[i] << 1;
            reg[i] ^= (qwIvReg >> (63 - i)) & 0x1;
        }
    for (int i = 0; i < 64; i++)
        for (int j = 0; j < 16; j++) {
            reg16[i] = reg16[i] << 1;
            reg16[i] ^= (qwIvReg >> (63 - i)) & 0x1;
        }
    for (int i = 0; i < 64; i++)
        for (int j = 0; j < 32; j++) {
            reg32[i] = reg32[i] << 1;
            reg32[i] ^= (qwIvReg >> (63 - i)) & 0x1;
        }       
    for (int t = 0; t < 10; t++)
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 8; j++) {
                bitsliced_keystream[t*8+i] = bitsliced_keystream[t*8+i] << 1;
                bitsliced_keystream[t*8+i] ^= (lpKsOut[t] >> (7 - i)) & 0x1;
            }
    for (int t = 0; t < 10; t++)
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 16; j++) {
                bitsliced_keystream16[t * 8 + i] = bitsliced_keystream16[t * 8 + i] << 1;
                bitsliced_keystream16[t * 8 + i] ^= (lpKsOut[t] >> (7 - i)) & 0x1;
            }
    for (int t = 0; t < 10; t++)
        for (int i = 0; i < 8; i++)
            for (int j = 0; j < 32; j++) {
                bitsliced_keystream32[t * 8 + i] = bitsliced_keystream32[t * 8 + i] << 1;
                bitsliced_keystream32[t * 8 + i] ^= (lpKsOut[t] >> (7 - i)) & 0x1;
            }
    hipMalloc((void**)&gpu_awTea3LutA, 8 * sizeof(uint16_t));
    hipMalloc((void**)&gpu_awTea3LutB, 8 * sizeof(uint16_t));
    hipMalloc((void**)&gpu_abTea3Sbox, 256 * sizeof(uint8_t));
    hipMalloc((void**)&gpu_lpKsOut, 10 * sizeof(uint8_t));
    hipMalloc((void**)&gpu_captured_key, 2 * sizeof(uint64_t));
    hipMalloc((void**)&bitsliced_keystream_d, 80 * sizeof(uint8_t));
    hipMalloc((void**)&bitsliced_keystream16_d, 80 * sizeof(uint16_t));
    hipMalloc((void**)&bitsliced_keystream32_d, 80 * sizeof(uint32_t));
    hipMalloc((void**)&reg_d, 64 * sizeof(uint8_t));
    hipMalloc((void**)&reg16_d, 64 * sizeof(uint16_t));
    hipMalloc((void**)&reg32_d, 64 * sizeof(uint32_t));

    hipMemcpy(gpu_awTea3LutA, g_awTea3LutA, 8 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_awTea3LutB, g_awTea3LutB, 8 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_abTea3Sbox, g_abTea3Sbox, 256 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_lpKsOut, lpKsOut, 10 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_captured_key, captured_key, 2 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(bitsliced_keystream_d, bitsliced_keystream, 80 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(bitsliced_keystream16_d, bitsliced_keystream16, 80 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(bitsliced_keystream32_d, bitsliced_keystream32, 80 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(reg_d, reg, 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(reg16_d, reg16, 64 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(reg32_d, reg32, 64 * sizeof(uint32_t), hipMemcpyHostToDevice);
    
    float time = 0;
    hipEvent_t startx, stopx;
    hipEventCreate(&startx);    hipEventCreate(&stopx);    hipEventRecord(startx);
    if (choice == 1) tea3_exhaustive << <BLOCKS, THREADS >> > (dwFrameNumbers, dwNumKsBytes, gpu_lpKsOut, gpu_awTea3LutA, gpu_awTea3LutB, gpu_abTea3Sbox, trials, gpu_captured_key);
    else if (choice == 2) tea3_exhaustive_0conflict << <BLOCKS, THREADS >> > (dwFrameNumbers, dwNumKsBytes, gpu_lpKsOut, gpu_awTea3LutA, gpu_awTea3LutB, gpu_abTea3Sbox, trials, gpu_captured_key);
    else if (choice == 3) tea3_exhaustive_1conflict << <BLOCKS, THREADS >> > (dwFrameNumbers, dwNumKsBytes, gpu_lpKsOut, gpu_awTea3LutA, gpu_awTea3LutB, gpu_abTea3Sbox, trials, gpu_captured_key);
    else if (choice == 11) tea3_exhaustive_bitsliced << <BLOCKS, THREADS >> > (reg_d, dwNumKsBytes, bitsliced_keystream_d, gpu_abTea3Sbox, trials, gpu_captured_key);
    else if (choice == 12) tea3_exhaustive_bitsliced_shared << <BLOCKS, THREADS >> > (reg_d, dwNumKsBytes, bitsliced_keystream_d, gpu_abTea3Sbox, trials, gpu_captured_key);
    else if (choice == 13) tea3_exhaustive_bitsliced16 << <BLOCKS, THREADS >> > (reg16_d, dwNumKsBytes, bitsliced_keystream16_d, gpu_abTea3Sbox, trials, gpu_captured_key);
    else if (choice == 14) tea3_exhaustive_bitsliced32 << <BLOCKS, THREADS >> > (reg32_d, dwNumKsBytes, bitsliced_keystream32_d, gpu_abTea3Sbox, trials, gpu_captured_key); // Best
    hipMemcpy(captured_key, gpu_captured_key, 2 * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipEventRecord(stopx);    hipEventSynchronize(stopx);    hipEventElapsedTime(&time, startx, stopx);
    printf("Captured key: %llx %llx\n", captured_key[0], captured_key[1]);
    printf("Elapsed time: %f\n", time);
    printf("%s\n", hipGetErrorString(hipGetLastError())); 
    return 0;
}